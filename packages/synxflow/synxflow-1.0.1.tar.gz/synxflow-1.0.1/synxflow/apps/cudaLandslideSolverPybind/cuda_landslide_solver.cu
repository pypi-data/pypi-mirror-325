#include "hip/hip_runtime.h"
// ====================================================================================== 
// Author              :    Xilin Xia, University of Birmingham, x.xia.1@bham.ac.uk
// Update Time         :    2023/09/19
// ======================================================================================
// LICENCE: GPLv3 
// ======================================================================================


/*!
\file cuda_debris_flow_solver.cu
\brief Source file for Debris Flow solver

*/
#ifdef _WIN32
#include <direct.h>
// MSDN recommends against using getcwd & chdir names
#define cwd _getcwd
#define cd _chdir
#else
#include "unistd.h"
#define cwd getcwd
#define cd chdir
#endif

#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//These header files are the primitive types
#include "Flag.h"
#include "Scalar.h"
#include "Vector.h"
#include "cuda_arrays.h"
//These header files are for the fields
#include "mapped_field.h"
#include "cuda_mapped_field.h"
//These header files are for finite volume mesh
#include "mesh_fv_reduced.h"
#include "mesh_fv_cartesian.h"
#include "mesh_interface.h"
#include "cuda_mesh_fv.h"
//These header files are for input and output
#include "gisAsciiMesh_reader.h"
#include "gmsh_reader.h"
#include "field_reader.h"
#include "cuda_simple_writer.h"
#include "cuda_backup_writer.h"
#include "cuda_gauges_writer.h"
#include "cuda_gisascii_writer.h"
//These header files are for shallow water equations advection
#include "cuda_advection_NSWEs.h"
//These header files are for shallow water equations advection with transport
#include "cuda_transport_NSWEs.h"
//erosion and deposition rate
#include "cuda_erosion_deposition.h"
//The header file for gradient
#include "cuda_gradient.h"
//The header file for hessian
#include "cuda_hessian.h"
//The header file for limiter
#include "cuda_limiter.h"
//The header file for friction
#include "cuda_friction.h"
//The header file for field algebra
#include "cuda_field_algebra.h"
//The header file for integrator
#include "cuda_integrators.h"
//The header file for device query
#include "cuda_device_query.h"
//The header file for time controllinh
#include "cuda_adaptive_time_control.h"
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <pybind11/pybind11.h>
namespace py = pybind11;

//using the name space for GeoClasses
using namespace GC;

int run(const char* work_dir){

  if(cd(work_dir) == -1){
    printf("The working directory does not exist!/n");
  }

  std::ifstream device_setup_file("input/device_setup.dat");
  int device_id;
  if (device_setup_file.is_open()){
    device_setup_file >> device_id;
	  checkCuda(hipSetDevice(device_id));
	  std::cout << "GPU " << device_id << " is choosen as the model device"<< std::endl;
  }
  else{
    deviceQuery();
  }

  //creating an structured finite volume mesh, it is initialized from DEM
  std::shared_ptr<unstructuredFvMesh>  mesh = std::make_shared<unstructuredReducedFvMesh>(gisAsciiMeshReader("input/mesh/DEM.txt"));

  std::cout<<"Read in mesh successfully"<<std::endl;

  //creating mesh on device
  std::shared_ptr<cuUnstructuredFvMesh>  mesh_ptr_dev = std::make_shared<cuUnstructuredFvMesh>(fvMeshQueries(mesh));
  
  //Read in field data
  fvScalarFieldOnCell z_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "z"));
  fvScalarFieldOnCell h_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "h"));
  fvVectorFieldOnCell hU_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "hU"));

  std::cout << "Read in field successfully" << std::endl;

  //h, z, hU
  cuFvMappedField<Scalar, on_cell> z(z_host,mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> h(h_host,mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> hU(hU_host, mesh_ptr_dev);

  //Velocity
  cuFvMappedField<Vector, on_cell> u(hU, partial);

  //x and y components of hU
  cuFvMappedField<Scalar, on_cell> hUx(h, partial);
  cuFvMappedField<Scalar, on_cell> hUy(h, partial);

  //advections
  cuFvMappedField<Scalar, on_cell> h_advection(h, partial);
  cuFvMappedField<Vector, on_cell> hU_advection(hU, partial);

  //friction force
  cuFvMappedField<Vector, on_cell> friction_force(hU, partial);

  //flux
  cuFvMappedField<Scalar, on_halffacet> h_flux(mesh_ptr_dev);
  cuFvMappedField<Vector, on_halffacet> hU_flux(mesh_ptr_dev);

  //old values
  cuFvMappedField<Scalar, on_cell> h_old(h, partial);
  cuFvMappedField<Vector, on_cell> hU_old(hU, partial);
  cuFvMappedField<Scalar, on_cell> h_advection_old(h_advection, partial);
  cuFvMappedField<Vector, on_cell> hU_advection_old(hU_advection, partial);

  //gradients
  cuFvMappedField<Vector, on_cell> h_grad(mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> z_grad(mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> eta_grad(mesh_ptr_dev);
  cuFvMappedField<Tensor, on_cell> u_grad(mesh_ptr_dev);

  //modifying factor for gravity
  cuFvMappedField<Scalar, on_cell> phi_b(mesh_ptr_dev);


  //calculate topography gradient
  fv::cuGradient(z, z_grad);
  fv::cuGradientLimiterCartesian(z, z_grad);

  //hessian
  cuFvMappedField<Tensor, on_cell> z_hessian(mesh_ptr_dev);
  fv::cuHessianCartesian2D(z, z_hessian);

  //gravity
  cuFvMappedField<Scalar, on_cell> gravity(h, partial);
  //setting gravity to single value 9.81
  fv::cuUnaryOn(gravity, [] __device__ (Scalar& a) -> Scalar{return 9.81;}); 

  //centrifugal
  cuFvMappedField<Scalar, on_cell> centrifugal(h, partial);

  //gross vertical acceleration
  cuFvMappedField<Scalar, on_cell> gross_acc(h, partial);

  //surface elevation eta
  cuFvMappedField<Scalar, on_cell> eta(h, partial);

  Scalar dt_out = 0.5;
  Scalar backup_interval = 0.0;
  Scalar backup_time = 0.0;
  Scalar t_current = 0.0;
  Scalar t_out = 0.0;
  Scalar t_all = 0.0;
  Scalar t_small = 1e-10;
  Flag rheology_type = 0;
  Flag gravity_correction_type = 0;
  bool curvature_on = true;
  bool filter_mass_flux = true;

  //creating uniform friction coefficients field
  cuFvMappedField<Scalar, on_cell> miu1(h, partial);
  cuFvMappedField<Scalar, on_cell> miu2(h, partial);
  cuFvMappedField<Scalar, on_cell> cohesion(h, partial);
  cuFvMappedField<Scalar, on_cell> rho(h, partial);

  Scalar _miu, _miu1, _miu2, _cohesion, _rho, _L, _beta, _U;


    //*******************Read times setup value from file
  std::ifstream times_setup_file("input/times_setup.dat");
  if (!times_setup_file) {
    std::cout << "Please input current time, total time, output time interval and backup interval" << std::endl;
    std::cin >> t_current >> t_all >> dt_out >> backup_interval;
  }
  else {
    Scalar _time;
    std::vector<Scalar> GPU_Time_Values;
    while (times_setup_file >> _time) {
      GPU_Time_Values.push_back(_time);
    }
    t_current = GPU_Time_Values[0];
    t_all = GPU_Time_Values[1];
    dt_out = GPU_Time_Values[2];
    backup_interval = GPU_Time_Values[3];
    std::cout << "Current time: " << t_current << "s" << std::endl;
    std::cout << "Total time: " << t_all << "s" << std::endl;
    std::cout << "Output time interval: " << dt_out << "s" << std::endl;
    std::cout << "Backup interval: " << backup_interval << "s" << std::endl;
  }
  //********************************

  //reading setup configuration
  std::ifstream setup;
  std::string line;
  std::string word;
  setup.open("input/setup.conf");
  if (!setup){
    std::cout << "error: unable to open input file: " << "input/setup.conf" << std::endl;
  }
  getline(setup, line);
  getline(setup, line);
  std::stringstream(line) >> rheology_type;
  switch (rheology_type){
  case 1:
    getline(setup, line);
    getline(setup, line);
    std::stringstream(line) >> _miu>> _cohesion>> _rho;
    printf("%f %f %f\n", _miu, _cohesion, _rho);
    if (_rho <= 1e-6){
      std::cout << "Fatal error: Density is 0!" << std::endl;
    }
    fv::cuUnaryOn(miu1, [=] __device__(Scalar& a) -> Scalar{ return _miu; });
    fv::cuUnaryOn(cohesion, [=] __device__(Scalar& a) -> Scalar{ return _cohesion; });
    fv::cuUnaryOn(rho, [=] __device__(Scalar& a) -> Scalar{ return _rho; });
    break;
  case 2:
    getline(setup, line);
    getline(setup, line);
    std::stringstream(line) >> _miu1 >> _miu2 >>_L >> _beta;
    fv::cuUnaryOn(miu1, [=] __device__(Scalar& a) -> Scalar{ return _miu1; });
    fv::cuUnaryOn(miu2, [=] __device__(Scalar& a) -> Scalar{ return _miu2; });
    break;
  case 3:
    getline(setup, line);
    getline(setup, line);
    std::stringstream(line) >> _miu1 >> _miu2 >> _U;
    fv::cuUnaryOn(miu1, [=] __device__(Scalar& a) -> Scalar{ return _miu1; });
    fv::cuUnaryOn(miu2, [=] __device__(Scalar& a) -> Scalar{ return _miu2; });
    break;
  }
  getline(setup, line);
  getline(setup, line);
  std::stringstream(line) >> gravity_correction_type;
  getline(setup, line);
  setup>>std::boolalpha >> curvature_on;
  getline(setup, line);
  getline(setup, line);
  setup >> std::boolalpha >> filter_mass_flux;
  getline(setup, line);
  setup.close();
  std::cout << "Read in configuration successfully" << std::endl;

  t_out = t_current + dt_out;
  backup_time = t_current + backup_interval;

  cuAdaptiveTimeControl2D time_controller(0.005, t_all, 0.5, t_current);

  //ascii raster writer
  cuGisAsciiWriter raster_writer("input/mesh/DEM.txt");

  //write the initial profile
  cuSimpleWriterLowPrecision(z, "z", time_controller.current());
  cuSimpleWriterLowPrecision(h, "h", time_controller.current());
  cuSimpleWriterLowPrecision(hU, "hU", time_controller.current());
  cuSimpleWriterLowPrecision(eta, "elev", time_controller.current());

  double total_runtime = 0.0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //print current time
  std::cout << time_controller.current() << std::endl;

  auto filter = [] __device__(Vector& a, Scalar& b) ->Vector{
    if (b <= 1e-6){
      return Vector(0.0);
    }
    else{
      return a;
    }
  };

  fv::cuBinaryOn(hU, h, filter);

  auto filter1 = [] __device__(Scalar& a) ->Scalar{
    if (a <= 1e-10){
      return 0.0;
    }
    else{
      return a;
    }
  };

  auto filter2 = [] __device__(Scalar& a, Vector& b) ->Scalar{
    if (sqrt(dot(b, b)) <= 1e-8){
      return 0.0;
    }
    else{
      return a;
    }
  };

  //Main loop
  do{

    hipEventRecord(start);

    //First step of Runger-Kutta--------------------------------------

    //calculate the surface elevation
    fv::cuBinary(h, z, eta, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return a + b;});

    //calculate the velocity
    auto divide = [] __device__ (Vector& a, Scalar& b) ->Vector{
      if(b >= 1e-10){
        return a/b;
      }else{
        return Vector(0.0);
      }
    };

    fv::cuBinary(hU, h, u, divide); 

    //store the old values, may need to be replaced by assining operators
    fv::cuBinaryOn(h_old, h, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return b;});
    fv::cuBinaryOn(hU_old, hU, [] __device__ (Vector& a, Vector& b) -> Vector{return b;});

    //calculate gradient
    fv::cuGradient(h, h_grad);
    fv::cuGradient(eta, eta_grad);
    fv::cuGradient(u, u_grad);

    //Limiting gradient
    fv::cuGradientLimiterCartesian(h, h_grad);
    fv::cuGradientLimiterCartesian(eta, eta_grad);
    fv::cuGradientLimiterCartesian(u, u_grad);
    
    switch (gravity_correction_type){
    case 0:
      break;
    case 1:
      //setting gravity to single value 9.81
      fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
      fv::cuBinary(z_grad, gravity, gravity, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });
      break;
    case 2:
      //calculating modifying factor
      fv::cuBinary(z_grad, eta_grad, phi_b, [] __device__(Vector& a, Vector& b) -> Scalar{ return fmax(0.0, fmin(1.0, 1 / (1.0 + dot(a, b)))); });
      //setting gravity to single value 9.81
      fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
      fv::cuBinary(phi_b, gravity, gravity, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a*b; });
      break;
    }


    //calculate centrifugal force
    if (curvature_on){
      fv::cuBinary(u, z_hessian, centrifugal, [] __device__(Vector& a, Tensor& b) -> Scalar{ return dot(dot(b, a), a); });
      fv::cuBinary(z_grad, centrifugal, centrifugal, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });
      fv::cuBinary(gravity, centrifugal, centrifugal, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return fmin((Scalar)10.0*a, fmax((Scalar)-0.9*a, b)); });
    }

    //calculate advection
    fv::cuAdvectionNSWEs2ndRobustCurv(gravity, centrifugal, h, z, u, h_grad, eta_grad, u_grad, h_advection, hU_advection);

    //multiply advection with -1
    fv::cuUnaryOn(h_advection, [] __device__ (Scalar& a) -> Scalar{return -1.0*a;});
    fv::cuUnaryOn(hU_advection, [] __device__ (Vector& a) -> Vector{return -1.0*a;});

    //integration
    fv::cuEulerIntegrator(h, h_advection, time_controller.dt(), time_controller.current());
    fv::cuEulerIntegrator(hU, hU_advection, time_controller.dt(), time_controller.current());

    //store the old values, may need to be replaced by assining operators
    fv::cuBinaryOn(h_advection_old, h_advection, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return b;});
    fv::cuBinaryOn(hU_advection_old, hU_advection, [] __device__ (Vector& a, Vector& b) -> Vector{return b;});

    fv::cuUnaryOn(h, filter1);

    //Second step of Runger-Kutta--------------------------------------


    //calculate the surface elevation
    fv::cuBinary(h, z, eta, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return a + b;});

    //calculate the velocity
    fv::cuBinary(hU, h, u, divide); 

    //calculate gradient
    fv::cuGradient(h, h_grad);
    fv::cuGradient(eta, eta_grad);
    fv::cuGradient(u, u_grad);

    //Limiting gradient
    fv::cuGradientLimiterCartesian(h, h_grad);
    fv::cuGradientLimiterCartesian(eta, eta_grad);
    fv::cuGradientLimiterCartesian(u, u_grad);

    switch (gravity_correction_type){
    case 0:
      break;
    case 1:
      //setting gravity to single value 9.81
      fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
      fv::cuBinary(z_grad, gravity, gravity, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });
      break;
    case 2:
      //calculating modifying factor
      fv::cuBinary(z_grad, eta_grad, phi_b, [] __device__(Vector& a, Vector& b) -> Scalar{ return fmax(0.0, fmin(1.0, 1 / (1.0 + dot(a, b)))); });
      //setting gravity to single value 9.81
      fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
      fv::cuBinary(phi_b, gravity, gravity, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a*b; });
      break;
    }

    if (curvature_on){
      //calculate centrifugal force
      fv::cuBinary(u, z_hessian, centrifugal, [] __device__(Vector& a, Tensor& b) -> Scalar{ return dot(dot(b, a), a); });
      fv::cuBinary(z_grad, centrifugal, centrifugal, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });
      fv::cuBinary(gravity, centrifugal, centrifugal, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return fmin((Scalar)10.0*a, fmax((Scalar)-0.9*a, b)); });
    }

    //calculate advection
    fv::cuAdvectionNSWEs2ndRobustCurv(gravity, centrifugal, h, z, u, h_grad, eta_grad, u_grad, h_advection, hU_advection);

    //multiply advection with -1
    fv::cuUnaryOn(h_advection, [] __device__ (Scalar& a) -> Scalar{return -1.0*a;});
    fv::cuUnaryOn(hU_advection, [] __device__ (Vector& a) -> Vector{return -1.0*a;});

    fv::cuEulerIntegrator(h_old, h_advection_old, time_controller.dt() / 2.0, time_controller.current());
    fv::cuEulerIntegrator(hU_old, hU_advection_old, time_controller.dt() / 2.0, time_controller.current());
    fv::cuEulerIntegrator(h_old, h_advection, time_controller.dt() / 2.0, time_controller.current());
    fv::cuEulerIntegrator(hU_old, hU_advection, time_controller.dt() / 2.0, time_controller.current());

    //Write updated values
    fv::cuBinaryOn(h, h_old, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return b; });
    fv::cuBinaryOn(hU, hU_old, [] __device__(Vector& a, Vector& b) -> Vector{ return b; });

    hU.update_boundary_values();
    hU.update_time(time_controller.current(), time_controller.dt());

    fv::cuBinaryOn(hU, h, filter);
    fv::cuUnaryOn(h, filter1);

    //add centrifugal force to gravity to obtain gross vertical acceleration
    fv::cuBinary(gravity, centrifugal, gross_acc, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a + b; });

    //Calculating friction, use old h for depth, new hU for direction
    switch (rheology_type){
    case 1:
      fv::cuFrictionMCPlastic(time_controller.dt(), gross_acc, miu1, cohesion, rho, h, hU, z_grad, friction_force);
      break;
    case 2:
      fv::cuFrictionMuI(time_controller.dt(), gross_acc, miu1, miu2, _beta, _L, h, hU, z_grad, friction_force);
      break;
    case 3:
      fv::cuFrictionLucas(time_controller.dt(), gross_acc, miu1, miu2, _U, h, hU, z_grad, friction_force);
      break;
    }
    fv::cuEulerIntegrator(hU, friction_force, time_controller.dt(), time_controller.current());


    h.update_boundary_values();
    h.update_time(time_controller.current(), time_controller.dt()); 


    //forwarding the time
    time_controller.forward();
    time_controller.updateByCFL(gravity,h,hU);

    //print current time
    std::cout << time_controller.current() << std::endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time = 0.0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_runtime += elapsed_time;

    if (time_controller.current() >= t_out - t_small){
      std::cout << "Writing output files" << std::endl;
      raster_writer.write(h, "h", t_out);
      fv::cuUnary(hU, hUx, [] __device__(Vector& a) -> Scalar{ return a.x; });
      fv::cuUnary(hU, hUy, [] __device__(Vector& a) -> Scalar{ return a.y; });
      raster_writer.write(hUx, "hUx", t_out);
      raster_writer.write(hUy, "hUy", t_out);
      t_out += dt_out;
    }

    if (time_controller.current() >= backup_time - t_small){
      std::cout << "Writing backup files" << std::endl;
      cuBackupWriter(h, "h_backup_", backup_time);
      cuBackupWriter(hU, "hU_backup_", backup_time);
      backup_time += backup_interval;
    }

  } while (!time_controller.is_end());

  std::cout << "Total runtime " << total_runtime << "ms" << std::endl;

  return 0;

}

