#include "hip/hip_runtime.h"
// ====================================================================================== 
// Author              :    Xilin Xia, University of Birmingham, x.xia.1@bham.ac.uk
// Update Time         :    2024/08/30
// ======================================================================================
// LICENCE: GPLv3 
// ======================================================================================


/*!
\file cuda_debris_flow_solver.cu
\brief Source file for Debris Flow solver

*/
#ifdef _WIN32
#include <direct.h>
// MSDN recommends against using getcwd & chdir names
#define cwd _getcwd
#define cd _chdir
#else
#include "unistd.h"
#define cwd getcwd
#define cd chdir
#endif

#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//These header files are the primitive types
#include "Flag.h"
#include "Scalar.h"
#include "Vector.h"
#include "cuda_arrays.h"
//These header files are for the fields
#include "mapped_field.h"
#include "cuda_mapped_field.h"
//These header files are for finite volume mesh
#include "mesh_fv_reduced.h"
#include "mesh_fv_cartesian.h"
#include "mesh_interface.h"
#include "cuda_mesh_fv.h"
//These header files are for input and output
#include "gisAsciiMesh_reader.h"
#include "gmsh_reader.h"
#include "field_reader.h"
#include "cuda_simple_writer.h"
#include "cuda_backup_writer.h"
#include "cuda_gauges_writer.h"
#include "cuda_gisascii_writer.h"
//These header files are for shallow water equations advection
#include "cuda_advection_NSWEs.h"
//These header files are for shallow water equations advection with transport
#include "cuda_transport_NSWEs.h"
//erosion and deposition rate
#include "cuda_erosion_deposition.h"
//The header file for gradient
#include "cuda_gradient.h"
//The header file for limiter
#include "cuda_limiter.h"
//The header file for friction
#include "cuda_friction.h"
//The header file for field algebra
#include "cuda_field_algebra.h"
//The header file for integrator
#include "cuda_integrators.h"
//The header file for device query
#include "cuda_device_query.h"
//The header file for time controllinh
#include "cuda_adaptive_time_control.h"
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <pybind11/pybind11.h>
namespace py = pybind11;

//using the name space for GeoClasses
using namespace GC;

int run(const char* work_dir){

  if(cd(work_dir) == -1){
    printf("The working directory does not exist!/n");
  }

  std::ifstream device_setup_file("input/device_setup.dat");
  int device_id;
  if (device_setup_file.is_open()){
    device_setup_file >> device_id;
	  checkCuda(hipSetDevice(device_id));
	  std::cout << "GPU " << device_id << " is choosen as the model device"<< std::endl;
  }
  else{
    deviceQuery();
  }

  Scalar dt_out = 0.5;
  Scalar backup_interval = 0.0;
  Scalar backup_time = 0.0;
  Scalar t_current = 0.0;
  Scalar t_out = 0.0;
  Scalar t_all = 0.0;
  Scalar t_small = 1e-10;

  //*******************Read times setup value from file
  std::ifstream times_setup_file("input/times_setup.dat");
  if (!times_setup_file) {
    std::cout << "Please input current time, total time, output time interval and backup interval" << std::endl;
    std::cin >> t_current >> t_all >> dt_out >> backup_interval;
  }
  else {
    Scalar _time;
    std::vector<Scalar> GPU_Time_Values;
    while (times_setup_file >> _time) {
      GPU_Time_Values.push_back(_time);
    }
    t_current = GPU_Time_Values[0];
    t_all = GPU_Time_Values[1];
    dt_out = GPU_Time_Values[2];
    backup_interval = GPU_Time_Values[3];
    std::cout << "Current time: " << t_current << "s" << std::endl;
    std::cout << "Total time: " << t_all << "s" << std::endl;
    std::cout << "Output time interval: " << dt_out << "s" << std::endl;
    std::cout << "Backup interval: " << backup_interval << "s" << std::endl;
  }
 //********************************
 //*******************Read device setup value from file

  Scalar rho_water = 1000;
  Scalar rho_solid = 1580;
  Scalar dim_mid = 0.0039;
  Scalar porosity = 0.42;
  Scalar critical_slope = 0.7;
  Scalar alpha = 1.0;
  Scalar beta = 1.0;
  //******************Read parametres from file
  std::ifstream parameters_file("input/parameters.dat");
  if (!parameters_file) {
    std::cout << "Please input water density (kg/m3), solid density (kg/m3), particle diameter (m), porosity, critical slope, alpha and beta" << std::endl;
    std::cin >> rho_water >> rho_solid >> dim_mid >> porosity >> critical_slope >> alpha >> beta;
  }
  else{
    Scalar _parameter;
    std::vector<Scalar> Parameter_Values;
    while (parameters_file >> _parameter) {
      Parameter_Values.push_back(_parameter);
    }
    rho_water = Parameter_Values[0];
    rho_solid = Parameter_Values[1];
    dim_mid = Parameter_Values[2];
    porosity = Parameter_Values[3];
    critical_slope = Parameter_Values[4];
    alpha = Parameter_Values[5];
    beta = Parameter_Values[6];
  }


  //*******************************
  //******************Read parameters from file

  std::shared_ptr<unstructuredFvMesh>  mesh = std::make_shared<CartesianFvMesh>("input/mesh/DEM.txt");

  std::cout << "Read in mesh successfully" << std::endl;

  //creating mesh on device
  std::shared_ptr<cuUnstructuredFvMesh>  mesh_ptr_dev = std::make_shared<cuUnstructuredFvMesh>(fvMeshQueries(mesh));

  //Read in field data
  fvScalarFieldOnCell z_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "z"));
  fvScalarFieldOnCell h_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "h"));
  fvVectorFieldOnCell hU_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "hU"));
  fvScalarFieldOnCell C_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "C"));
  fvScalarFieldOnCell erodible_depth_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "erodible_depth"));
  fvScalarFieldOnCell manning_coeff_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "manning"));
  fvScalarFieldOnCell mu_dynamic_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "dynamic_friction_coeff"));
  fvScalarFieldOnCell mu_static_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "static_friction_coeff"));

  //precipitation
  fvScalarFieldOnCell precipitation_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "precipitation"));

  std::cout << "Read in field successfully" << std::endl;

  //h, z, hU, C
  cuFvMappedField<Scalar, on_cell> z_old(z_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> z(z_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> h(h_host, mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> hU(hU_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> C(C_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> erodible_depth(erodible_depth_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> manning_coeff(manning_coeff_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> mu_dynamic(mu_dynamic_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> mu_static(mu_static_host, mesh_ptr_dev);

  //precipitation
  cuFvMappedField<Scalar, on_cell> precipitation(precipitation_host, mesh_ptr_dev);

  //new topography after collapse
  cuFvMappedField<Scalar, on_cell> z_new(h, partial);

  //Velocity
  cuFvMappedField<Vector, on_cell> u(hU, partial);

  //x and y components of hU
  cuFvMappedField<Scalar, on_cell> hUx(h, partial);
  cuFvMappedField<Scalar, on_cell> hUy(h, partial);

  //Solid phase depth
  cuFvMappedField<Scalar, on_cell> hC(h, partial);

  //concentration gradient
  cuFvMappedField<Vector, on_cell> C_grad(hU, partial);

  //creating gauges writer
  cuGaugesWriter<Scalar, on_cell> h_writer(fvMeshQueries(mesh), h, "input/field/gauges_pos.dat", "output/h_gauges.dat");
  cuGaugesWriter<Vector, on_cell> hU_writer(fvMeshQueries(mesh), hU, "input/field/gauges_pos.dat", "output/hU_gauges.dat");
  cuGaugesWriter<Scalar, on_cell> C_writer(fvMeshQueries(mesh), C, "input/field/gauges_pos.dat", "output/C_gauges.dat");

  //----Below is not needed------
  //find minimum z
  //Scalar min_z = thrust::reduce(thrust::device_ptr <Scalar>(z.data.dev_ptr()), thrust::device_ptr <Scalar>(z.data.dev_ptr() + z.data.size()), (Scalar) 3e35, thrust::minimum<Scalar>());

  //fv::cuUnaryOn(z, [=] __device__(Scalar& a) -> Scalar{ return a - min_z + 0.0001; });
  //-----------------------------
  //advections
  cuFvMappedField<Scalar, on_cell> h_advection(h, partial);
  cuFvMappedField<Vector, on_cell> hU_advection(hU, partial);
  cuFvMappedField<Scalar, on_cell> hC_advection(hC, partial);

  //friction force
  cuFvMappedField<Vector, on_cell> friction_force(hU, partial);

  //erosion and deposition rate
  cuFvMappedField<Scalar, on_cell> ED_rate(h, partial);

  //erosion and deposition momentum correction
  cuFvMappedField<Vector, on_cell> mom_correction(hU, partial);

  //gradient
  cuFvMappedField<Vector, on_cell> z_gradient(hU, partial);
  fv::cuLimitedGradientCartesian(z, z_gradient);

  //mass positivity time step constrain
//  cuFvMappedField<Scalar, on_cell> dt_mass(h, partial);

  //gravity
  cuFvMappedField<Scalar, on_cell> gravity(h, partial);
  //setting gravity to single value 9.81
  fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
  //fv::cuBinary(z_gradient, gravity, gravity, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });

  t_out = t_current + dt_out;
  backup_time = t_current + backup_interval;

  cuAdaptiveTimeControl2D time_controller(0.005, t_all, 0.5, t_current);

  //write the initial profile
  //std::cout << "Writing initial conditions as output" << std::endl;
  //cuSimpleWriterLowPrecision(z, "z", time_controller.current());
  //cuSimpleWriterLowPrecision(h, "h", time_controller.current());
  //cuSimpleWriterLowPrecision(hU, "hU", time_controller.current());
  //cuSimpleWriterLowPrecision(hC, "hC", time_controller.current());

  auto filter = [] __device__(Vector& a, Scalar& b) ->Vector{
    if (b <= 1e-10){
      return Vector(0.0);
    }
    else{
      return a;
    }
  };

  auto filter2 = [] __device__(Scalar& a, Scalar& b) ->Scalar{
    if (a <= 1e-10){
      return 0.0;
    }
    else if(a >= b){
      return b;
    }else{
      return a;
    }
  };

  auto filter1 = [] __device__(Scalar& a) ->Scalar{
    if (a <= 1e-10){
      return 0.0;
    }
    else{
      return a;
    }
  };

  //calculate the velocity
  auto divide = [] __device__(Vector& a, Scalar& b) ->Vector{
    if (b >= 1e-10){
      return a / b;
    }
    else{
      return Vector(0.0);
    }
  };

  auto divide_scalar = [] __device__(Scalar& a, Scalar& b) ->Scalar{
    if (b >= 1e-10){
      return a / b;
    }
    else{
      return Scalar(0.0);
    }
  };

  auto multiply_scalar = [] __device__(Scalar& a, Scalar& b) ->Scalar{
    return a*b;
  };  

  //print current time
  std::cout << time_controller.current() << std::endl;

  z.update_time(time_controller.current(), 0.0);
  z.update_boundary_values();
  h.update_time(time_controller.current(), 0.0);
  h.update_boundary_values();
  hU.update_time(time_controller.current(), 0.0);
  hU.update_boundary_values();
  C.update_time(time_controller.current(), 0.0);
  C.update_boundary_values();
  
  //hC for the initial step
  fv::cuBinary(h, C, hC, multiply_scalar);

  //ascii raster writer
  cuGisAsciiWriter raster_writer("input/mesh/DEM.txt");

  //write initial depth
  raster_writer.write(h, "h", time_controller.current());

  std::ofstream fout;
  fout.open("output/timestep_log.txt");

  double total_runtime = 0.0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  int cnt = 0;
  //Main loop
  do{

    hipEventRecord(start);

    Scalar dt = time_controller.dt();

    //calculate the surface elevation
    fv::cuBinary(hU, h, u, divide);

    //calculate advection
    fv::cuTransportNSWEsSRMCartesian(gravity, h, z, z_gradient, hU, hC, h_advection, hU_advection, hC_advection);

    //multiply advection with -1
    fv::cuUnaryOn(h_advection, [] __device__(Scalar& a) -> Scalar{ return -1.0*a; });
    fv::cuUnaryOn(hU_advection, [] __device__(Vector& a) -> Vector{ return -1.0*a; });
    fv::cuUnaryOn(hC_advection, [] __device__(Scalar& a) -> Scalar{ return -1.0*a; });

    //integration
    fv::cuEulerIntegrator(h, h_advection, time_controller.dt(), time_controller.current());
    fv::cuEulerIntegrator(hC, hC_advection, time_controller.dt(), time_controller.current());
    //fv::cuFrictionManningMCImplicit(time_controller.dt(), porosity, rho_water, rho_solid, gravity, manning_coeff, mu_dynamic, h, hC, hU, hU_advection);
    fv::cuFrictionManningImplicit(time_controller.dt(), gravity, manning_coeff, h, hU, hU_advection);

    //calculating erosion deposition rate
    //fv::cuEDMeyerPeterMuller(time_controller.dt(), gravity, h, hU, hC, manning_coeff, ED_rate, rho_solid, rho_water, dim_mid);
    fv::cuEDTakahashiIversonXia(gravity, h, hU, hC, manning_coeff, mu_dynamic, mu_static, ED_rate, rho_solid, rho_water, porosity, alpha, beta, dt, dim_mid);

    hU.update_time(time_controller.current(), time_controller.dt());
    hU.update_boundary_values();

    //constraining erosion deposition rate
    fv::cuBinary(ED_rate, hC, ED_rate, [=] __device__(Scalar& a, Scalar& b) -> Scalar{ return fmax(a, (Scalar)-1.0*b / dt); });
    //fv::cuBinary(ED_rate, hC, ED_rate, [=] __device__(Scalar& a, Scalar& b) -> Scalar { return fmax(a, 0.0); });
    fv::cuBinary(ED_rate, erodible_depth, ED_rate, [=] __device__(Scalar& a, Scalar& b) -> Scalar{ return fmin(a, b*((Scalar)1.0 - porosity) / dt); });
    
    //calculate the concentration
    fv::cuBinary(hC, h, C, divide_scalar);

    //calculating the concentration gradient
    fv::cuLimitedGradientCartesian(C, C_grad);

    //calculating momentum correction term
    fv::cuMomentumCorrection(gravity, h, hC, C_grad, hU, ED_rate, mom_correction, rho_solid, rho_water, porosity);

    //updating momentum
    fv::cuEulerIntegrator(hU, mom_correction, time_controller.dt(), time_controller.current());

    //updating solid phase depth
    fv::cuEulerIntegrator(hC, ED_rate, time_controller.dt(), time_controller.current());

    fv::cuUnaryOn(ED_rate, [=] __device__(Scalar& a) -> Scalar{ return a / (1.0 - porosity); });

    //updating depth
    fv::cuEulerIntegrator(h, ED_rate, time_controller.dt(), time_controller.current());

    //updating bed 
    fv::cuEulerIntegrator(z, ED_rate, -1.0*time_controller.dt(), time_controller.current());

    //updating erodible depth 
    fv::cuEulerIntegrator(erodible_depth, ED_rate, -1.0*time_controller.dt(), time_controller.current());

    //bank collpse
    //  if (cnt % 100 == 0){
    //   for (int i = 0; i < 10; i++){
    //     fv::cuBankCollapse(z, z_new, critical_slope);
    //     fv::cuUnary(z_new, z, [=] __device__(Scalar& a) -> Scalar{ return a; });
    //   }
    // }

    //precipitation
    precipitation.update_time(time_controller.current(), 0.0);
    precipitation.update_data_values();
    fv::cuEulerIntegrator(h, precipitation, time_controller.dt(), time_controller.current());

    //calculate the concentration, update the boundary, and then calculate hC again, so that concentration boundary condition can be applied
    fv::cuBinary(hC, h, C, divide_scalar);
    C.update_time(time_controller.current(), time_controller.dt());
    C.update_boundary_values();
    fv::cuBinary(h, C, hC, multiply_scalar);


    //forwarding the time
    time_controller.forward();
    time_controller.updateByCFL(gravity, h, hU);
// xiaodong add this to reduce the size of output gauge files
    if (cnt % 100 == 0){
        h_writer.write(time_controller.current());
        hU_writer.write(time_controller.current());
        C_writer.write(time_controller.current());
    }
//
    //h_writer.write(time_controller.current());
    //hU_writer.write(time_controller.current());

    //print current time
    printf("%f\n", time_controller.current());
    fout << time_controller.current() << " " << time_controller.dt() << std::endl;
    cnt++;

    fv::cuBinaryOn(hU, h, filter);
    //fv::cuUnaryOn(h, filter1);
    //fv::cuBinaryOn(hC, h, filter2);


    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time = 0.0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_runtime += elapsed_time;

    if (time_controller.current() >= t_out - t_small){
      std::cout << "Writing output files" << std::endl;
      raster_writer.write(z, "z", t_out);
      raster_writer.write(h, "h", t_out);
      fv::cuUnary(hU, hUx, [] __device__(Vector& a) -> Scalar{ return a.x; });
      fv::cuUnary(hU, hUy, [] __device__(Vector& a) -> Scalar{ return a.y; });
      raster_writer.write(hUx, "hUx", t_out);
      raster_writer.write(hUy, "hUy", t_out);
      raster_writer.write(C, "C", t_out);
      t_out += dt_out;
    }

    if (time_controller.current() >= backup_time - t_small){
      std::cout << "Writing backup files" << std::endl;
      cuBackupWriter(h, "h_backup_", backup_time);
      cuBackupWriter(z, "z_backup_", backup_time);
      cuBackupWriter(hU, "hU_backup_", backup_time);
      cuBackupWriter(C, "C_backup_", backup_time);
      backup_time += backup_interval;
    }



  } while (!time_controller.is_end());

  std::cout << "Total runtime " << total_runtime << "ms" << std::endl;

  return 0;

}

