#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "../chess_consts.h"
#include "../clamp.cu"

__device__ bool bishop_move(
    size_t env,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions
) {
    // performs a bishop movement
    // returns 0 if the action was performed
    // returns 1 if the action was not applicable
    // this routine does not verify if the bishop is in check
    
    const unsigned char player_bishop = players[env] * 6 + WHITE_BISHOP;
    const unsigned char source = actions[0][env] * 8 + actions[1][env];
    const unsigned char target = actions[2][env] * 8 + actions[3][env];
    const unsigned char enemy_pawn  = ((players[env] + 1) % 2) * 6 + WHITE_PAWN;
    const unsigned char enemy_queen = ((players[env] + 1) % 2) * 6 + WHITE_QUEEN;
    const unsigned char srcrow = actions[0][env];
    const unsigned char srccol = actions[1][env];
    const unsigned char tgtrow = actions[2][env];
    const unsigned char tgtcol = actions[3][env];

    const char dir_x = (+1) * (tgtcol > srccol) + (-1) * (tgtcol < srccol);
    const char dir_y = (+1) * (tgtrow > srcrow) + (-1) * (tgtrow < srcrow);
    bool is_jumping_over = false;
    for (int i = 1; i < abs(tgtcol - srccol); i++) {
        is_jumping_over = is_jumping_over | (boards[clamp(0,63,(srcrow + i * dir_y) * 8 + (srccol + i * dir_x))][env] != EMPTY);
    }

    const bool is_action_ok = (
        (actions[4][env] == 0)               &   // no special action
        (boards[source][env] == player_bishop) & ( // source is a bishop
            (abs(srcrow - tgtrow) == abs(srccol - tgtcol)) // bishop moving diagonally
        ) & 
        !is_jumping_over & // bishop is not jumping over other pieces
        ( // target is a valid bishop movement
            (boards[target][env] == EMPTY) |
            ((boards[target][env] >= enemy_pawn) & (boards[target][env] <= enemy_queen))
        ) // target is empty or enemy
    );

    boards[target][env] = is_action_ok ? player_bishop : boards[target][env];
    boards[source][env] = is_action_ok ? EMPTY         : boards[source][env];

    return !is_action_ok;
}

__global__ void bishop_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> result
) {
    const int env = blockIdx.x * blockDim.x + threadIdx.x;
    if (env < boards.size(1)) result[env] = bishop_move(env, players, boards, actions);
}


