#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "../chess_consts.h"

__device__ bool rook_move(
    size_t env,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions
) {
    // performs a rook movement
    // returns 0 if the action was performed
    // returns 1 if the action was not applicable
    // this routine does not verify if the rook is in check
    
    const unsigned char player_rook = players[env] * 6 + WHITE_ROOK;
    const unsigned char source = actions[0][env] * 8 + actions[1][env];
    const unsigned char target = actions[2][env] * 8 + actions[3][env];
    const unsigned char enemy_pawn  = ((players[env] + 1) % 2) * 6 + WHITE_PAWN;
    const unsigned char enemy_queen = ((players[env] + 1) % 2) * 6 + WHITE_QUEEN;
    const unsigned char srcrow = actions[0][env];
    const unsigned char srccol = actions[1][env];
    const unsigned char tgtrow = actions[2][env];
    const unsigned char tgtcol = actions[3][env];

    const char dir_x = (+1) * (srccol < tgtcol) + (-1) * (srccol > tgtcol);
    const char dir_y = (+1) * (srcrow < tgtrow) + (-1) * (srcrow > tgtrow);
    bool is_jumping_over = false;
    for (int i = 1; i < abs(tgtcol - srccol) * (srcrow == tgtrow) + abs(tgtrow - srcrow) * (srccol==tgtcol); i++) {
        is_jumping_over = is_jumping_over | (boards[(srcrow + i * dir_y) * 8 + (srccol + i * dir_x)][env] != EMPTY);
    }

    const bool is_action_ok = (
        (actions[4][env] == 0)               & // no special action
        (boards[source][env] == player_rook) & // source is a rook
        !is_jumping_over & (                   // rook is not jumping over other pieces
            ((srcrow == tgtrow) & (srccol <= 7)) |
            ((srccol == tgtcol) & (srcrow <= 7))
        ) & ( // target is a valid rook movement
            (boards[target][env] == EMPTY) |
            ((boards[target][env] >= enemy_pawn) & (boards[target][env] <= enemy_queen))
        ) // target is empty or enemy
    );

    boards[target][env] = is_action_ok ? player_rook : boards[target][env];
    boards[source][env] = is_action_ok ? EMPTY       : boards[source][env];

    return !is_action_ok;
}

__global__ void rook_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> result
) {
    const int env = blockIdx.x * blockDim.x + threadIdx.x;
    if (env < boards.size(1)) result[env] = rook_move(env, players, boards, actions);
}


