#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "../chess_consts.h"
#include "../clamp.cu"

__device__ bool queen_move(
    size_t env,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions
) {
    // performs a queen movement
    // returns 0 if the action was performed
    // returns 1 if the action was not applicable
    // this routine does not verify if the queen is in check
    
    const unsigned char player_queen = players[env] * 6 + WHITE_QUEEN;
    const unsigned char source = actions[0][env] * 8 + actions[1][env];
    const unsigned char target = actions[2][env] * 8 + actions[3][env];
    const unsigned char enemy_pawn  = ((players[env] + 1) % 2) * 6 + WHITE_PAWN;
    const unsigned char enemy_queen = ((players[env] + 1) % 2) * 6 + WHITE_QUEEN;
    const unsigned char srcrow = actions[0][env];
    const unsigned char srccol = actions[1][env];
    const unsigned char tgtrow = actions[2][env];
    const unsigned char tgtcol = actions[3][env];

    const char dir_x = (+1) * (srccol < tgtcol) + (-1) * (srccol > tgtcol);
    const char dir_y = (+1) * (srcrow < tgtrow) + (-1) * (srcrow > tgtrow);
    bool is_jumping_over = false;
    bool encountered_target = false;
    for (int i = 1; i < 8; i++) {
        encountered_target = encountered_target | ((srcrow + i * dir_y == tgtrow) & (srccol + i * dir_x == tgtcol));
        is_jumping_over = is_jumping_over | ((!encountered_target) & (boards[clamp(0,63,(srcrow + i * dir_y) * 8 + (srccol + i * dir_x))][env] != EMPTY));
    }

    const bool is_action_ok = (
        (actions[4][env] == 0)                & // no special action
        (boards[source][env] == player_queen) & // source is a queen
        !is_jumping_over & (                    // queen is not jumping over other pieces
            ((srcrow == tgtrow) & (srccol <= 7)) |
            ((srccol == tgtcol) & (srcrow <= 7)) |
            (abs(srcrow - tgtrow) == abs(srccol - tgtcol))
        ) & ( // target is a valid queen movement
            (boards[target][env] == EMPTY) |
            ((boards[target][env] >= enemy_pawn) & (boards[target][env] <= enemy_queen))
        ) // target is empty or enemy
    );

    boards[target][env] = is_action_ok ? player_queen : boards[target][env];
    boards[source][env] = is_action_ok ? EMPTY       : boards[source][env];

    return !is_action_ok;
}

__global__ void queen_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> result
) {
    const int env = blockIdx.x * blockDim.x + threadIdx.x;
    if (env < boards.size(1)) result[env] = queen_move(env, players, boards, actions);
}


